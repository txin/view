
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 512

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int *a) {
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 10;
    } 
}


int main(void) {
    int *a, *b, *c;
//    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    // int size = sizeof(int);
    
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    a = (int *)malloc(size); random_ints(a);
    b = (int *)malloc(size); random_ints(b);
    c = (int *)malloc(size);
    
    //a = 2;
    //b = 7;

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // add<<<1, 1>>>(d_a, d_b, d_c);
    add<<<N, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    //printf("c=%d", c);
    for (int i = 0; i < N; i++) {
        printf("%d", c[i]);
    }
    
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}
