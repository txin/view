
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (2048 * 2048)
#define M 512

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void random_ints(int *a) {
    for (int i = 0; i < N; i++) {
        a[i] = rand() % 10;
    } 
}


int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
        
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    a = (int *)malloc(size); random_ints(a);
    b = (int *)malloc(size); random_ints(b);
    c = (int *)malloc(size);
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    add<<<(N + M - 1) / M, M>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    
    // for (int i = 0; i < N; i++) {
    //    printf("%d", c[i]);
    //}
    
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}
